
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10000


void initializeMatrices(int a[N][N], int b[N][N]) {
	srand(time(NULL));
	for (int i=0; i<N; i++) {
		for (int j=0; j<N; j++) {
			a[i][j] = rand() % 50;
			b[i][j] = rand() % 50;
		}
	}
}

__global__ void matrixProduct(int *a, int *b, int *c, int width) {
	int sum = 0;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	// printf("Thread in block position: (%d, %d) \n", row, col);
	if (col < width && row < width) {
		for (int k=0; k<width; k++) {
			sum += a[row * width + k] * b[k * width + col];
		}
		c[row * width + col] = sum;
	}
}

void showMatrices(int a[N][N], int b[N][N], int c[N][N]) {
	printf("***** MATRIX A *****\n");
	for (int i=0; i<N; i++) {
		for (int j=0; j<N; j++) {
			(j % N == N-1) ? printf("%d \n", a[i][j]) : printf("%d,", a[i][j]);
		}
	}
	printf("***** MATRIX B *****\n");
	for (int i=0; i<N; i++) {
		for (int j=0; j<N; j++) {
			(j % N == N-1) ? printf("%d \n", b[i][j]) : printf("%d,", b[i][j]);
		}
	}
	printf("***** MATRIX C *****\n");
	for (int i=0; i<N; i++) {
		for (int j=0; j<N; j++) {
			(j % N == N-1) ? printf("%d \n", c[i][j]) : printf("%d,", c[i][j]);
		}
	}
}

int main() {
	struct timeval t1, t2;
	gettimeofday(&t1, 0);

	int h_a[N][N], h_b[N][N], h_c[N][N];
	int *d_a, *d_b, *d_c;

	initializeMatrices(h_a, h_b);

	double size = (double) N * N * sizeof(int);
	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);

	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

	dim3 dimGrid(1, 1);
	dim3 dimBlock(N, N);

	matrixProduct<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);
	hipDeviceSynchronize();
	hipGetLastError();

	hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	// showMatrices(a, b, c);

	hipDeviceReset();

	gettimeofday(&t2, 0);
	double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
	printf("Time to calculate:  %3.1f ms \n", time);
	
	return 0;
}
